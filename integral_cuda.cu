#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <iomanip>
#include <time.h>
#include <math.h>
#include <chrono>
#include <unistd.h>

#define DO_PRINT false

double func(auto x);

double calc_integral(auto seg_count, auto a, auto b);
double calc_integral_with_accuracy(auto a, auto b, auto accuracy, auto init_segments_count);
double calc_runge_error(auto sum_prev_step, auto sum_cur_step);

void print_step(auto iter, auto seg_count, auto sum, auto err_runge);
void print_step_winner(auto iter, auto seg_count, auto sum, auto err_runge);
void print_header();
void print_footer();

int main()
{
    auto a = 1.0;
    auto b = 2.0;
    auto accuracy = 1E-50;
    auto init_segments_count = 100000;

    auto start = std::chrono::high_resolution_clock::now();
    auto result = calc_integral_with_accuracy(a, b, accuracy, init_segments_count);
    auto end = std::chrono::high_resolution_clock::now();

    std::cout << "result: " << std::fixed << std::setprecision(30) << result << std::endl;
    std::cout << "time: " << std::chrono::duration_cast<std::chrono::microseconds>(end - start).count() << " microseconds" << std::endl;
}

double calc_runge_error(auto sum_prev_step, auto sum_cur_step)
{
    return abs(sum_cur_step - sum_prev_step);
}

//device func
__device__ double func_device(double x)
{
    return 2 - x + cos(x) - log10(1 + x);
}

// cuda kernal calc_integral
__global__ void calc_integral_kernel(double *a_d, double *b_d, double *step_size_d, double *sum_d, int seg_count)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < seg_count)
    {
        double x_left = a_d[0] + i * step_size_d[0];
        double x_right = a_d[0] + (i + 1) * step_size_d[0];
        sum_d[i] = ((x_right - x_left) / 6) * (func_device(x_left) + 4 * func_device((x_left + x_right) / 2) + func_device(x_right));
    }
}

double calc_integral_with_accuracy(auto a, auto b, auto accuracy, auto init_segments_count)
{
    print_header();

    auto err_runge = 1.0;
    auto seg_count = init_segments_count;
    auto iter = 1;

    // allocate memory on host
    double *a_h = (double *)malloc(sizeof(double));
    double *b_h = (double *)malloc(sizeof(double));
    double *step_size_h = (double *)malloc(sizeof(double));
    double *sum_h = (double *)malloc(sizeof(double) * seg_count);

    // allocate memory on device
    double *a_d;
    double *b_d;
    double *step_size_d;
    double *sum_d;
    hipMalloc((void **)&a_d, sizeof(double));
    hipMalloc((void **)&b_d, sizeof(double));
    hipMalloc((void **)&step_size_d, sizeof(double));
    hipMalloc((void **)&sum_d, sizeof(double) * seg_count);

    // copy data from host to device
    a_h[0] = a;
    b_h[0] = b;
    hipMemcpy(a_d, a_h, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(b_d, b_h, sizeof(double), hipMemcpyHostToDevice);

    // calculate step size
    step_size_h[0] = (b - a) / seg_count;
    hipMemcpy(step_size_d, step_size_h, sizeof(double), hipMemcpyHostToDevice);

    // calculate sum
    calc_integral_kernel<<<(seg_count + 255) / 256, 256>>>(a_d, b_d, step_size_d, sum_d, seg_count);
    hipMemcpy(sum_h, sum_d, sizeof(double) * seg_count, hipMemcpyDeviceToHost);

    // sum up the result
    auto sum_prev_step = 0.0;
    for (auto i = 0; i < seg_count; i++)
    {
        sum_prev_step += sum_h[i];
    }

    seg_count += 2;

    while (err_runge > accuracy)
    {
        print_step(iter, seg_count, sum_prev_step, err_runge);

        // calculate step size
        step_size_h[0] = (b - a) / seg_count;
        hipMemcpy(step_size_d, step_size_h, sizeof(double), hipMemcpyHostToDevice);

        // calculate sum
        calc_integral_kernel<<<(seg_count + 255) / 256, 256>>>(a_d, b_d, step_size_d, sum_d, seg_count);
        hipMemcpy(sum_h, sum_d, sizeof(double) * seg_count, hipMemcpyDeviceToHost);

        // sum up the result
        auto sum_cur_step = 0.0;
        for (auto i = 0; i < seg_count; i++)
        {
            sum_cur_step += sum_h[i];
        }

        seg_count += 2;
        err_runge = calc_runge_error(sum_prev_step, sum_cur_step);
        sum_prev_step = sum_cur_step;
        iter++;
    }

    print_step_winner(iter, seg_count, sum_prev_step, err_runge);
    print_footer();

    // free memory on device
    hipFree(a_d);
    hipFree(b_d);
    hipFree(step_size_d);
    hipFree(sum_d);

    // free memory on host
    free(a_h);
    free(b_h);
    free(step_size_h);
    free(sum_h);

    return sum_prev_step;
}

void print_step(auto iter, auto seg_count, auto sum, auto err_runge)
{
    if (!DO_PRINT)
        return;
    std::cout << std::setw(5) << iter << " | ";
    std::cout << std::setw(9) << seg_count << " | ";
    std::cout << std::setw(15) << std::setprecision(15) << sum << " | ";
    std::cout << std::setw(8) << std::setprecision(3) << err_runge << std::endl;
}

void print_step_winner(auto iter, auto seg_count, auto sum, auto err_runge)
{
    if (!DO_PRINT)
        return;
    std::cout << std::setw(5) << iter << " | ";
    std::cout << std::setw(9) << seg_count << " | ";
    std::cout << std::setw(15) << std::setprecision(15) << sum << " | ";
    std::cout << std::setw(8) << std::setprecision(3) << err_runge;
    std::cout << " <--- winner" << std::endl;
}

void print_header()
{
    if (!DO_PRINT)
        return;
    std::cout << std::setw(5) << "iter"
              << " | ";
    std::cout << std::setw(9) << "seg_count"
              << " | ";
    std::cout << std::setw(17) << "sum"
              << " | ";
    std::cout << std::setw(8) << "err_runge" << std::endl;

    std::cout << std::setw(5) << "----"
              << " | ";
    std::cout << std::setw(9) << "---------"
              << " | ";
    std::cout << std::setw(17) << "-----------------"
              << " | ";
    std::cout << std::setw(8) << "---------" << std::endl;
}

void print_footer()
{
    if (!DO_PRINT)
        return;
    std::cout << std::setw(5) << "----"
              << " | ";
    std::cout << std::setw(9) << "---------"
              << " | ";
    std::cout << std::setw(17) << "-----------------"
              << " | ";
    std::cout << std::setw(8) << "---------" << std::endl;
}
